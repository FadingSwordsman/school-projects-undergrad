#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define BLOCK_SIZE
#define NUM_BLOCKS

__device__ int *ciphertext;

int *plaintext;

__global__ void calculate_entropy()
{
	
}

__global__ void run_cipher()
{
	
} 

__global__ setup_gpu()
{
	
}

int main(int argc, char **argv)
{
	int x;
	char* ciphertext;
	setup_gpu();

	ciphertext

	for(x = 0; x < BLOCK_SIZE*NUM_BLOCKS; x += BLOCK_SIZE)
	{
		//Thank you http://on-demand.gputechconf.com/gtc-express/2011/presentations/StreamsAndConcurrencyWebinar.pdf
		run_cipher <<< block, 0, stream2 >>> ();
		
		calculate_entropy <<< block, NUM_THREADS, 0, stream3 >>> ();
	}
}
